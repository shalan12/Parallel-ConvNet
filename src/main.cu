#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstddef>
#include <iostream>
#include <numeric>
#include <map>
#include <sys/time.h>
#include <valarray>

#include <hdf5.h>

#include "range.hpp"
#include "utils.hpp"

#define NUM_ROWS 28
#define NUM_COLS 28
#define NUM_CHANNELS 1
#define NUM_DIGITS 10
#define TILE_SIZE 8
#define MperBlock 16
#define FILTER_SIZE 5 

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      printf("Failed to run stmt %s\n", #stmt);                         \
      printf("ERROR - Got CUDA error ...  %s\n", hipGetErrorString(err));      \
      return ;                                                            \
    }                                                                     \
  } while (0)

static int FLAGS_batch_size = 10000; // number of images....actual value changes at runtime
static std::string FLAGS_testdata{};
static std::string FLAGS_model{};

// Data and reference data dimensions
static int xdims[] = {FLAGS_batch_size, NUM_ROWS, NUM_COLS, NUM_CHANNELS};
static int rdims[] = {FLAGS_batch_size, NUM_DIGITS};

// Model dimensions
static int conv1dims[] = {5, 5, 1, 32}; // rows, cols, #input_feature maps, #output_feature_maps
static int conv2dims[] = {5, 5, 32, 64}; // rows, cols, #input_feature maps, #output_feature_maps
static int fc1dims[]   = {1024, 128}; // not important for convolution or subsampling layers
static int fc2dims[]   = {128, 10}; // not important for convolution or subsampling layers

__constant__ float mask1[5][5][32];

void convolveWrapper(const float *X, const int xdims[4], const float *W, const int wdims[4], float *Y,
                                  const int ydims[4], bool useConstMemory=false);
__global__ void convolve(const float *X, const int xdims[4], const float *W, const int wdims[4], float *Y,
                         const int ydims[4], int W_grid, int n);
__global__ void convolve1(const float *X, const int xdims[4], float *Y, const int ydims[4], int W_grid, int num_images);

void easyConvWrapper (const float *X, const int xdims[4], const float *W, const int wdims[4], float *Y, const int ydims[4]);
__global__ void easyConv (const float *X, const int xdims[4], const float *W, const int wdims[4], float *Y,
                          const int ydims[4], int* W_grid);

static int loadData(float *x, float *y) {
  // Open the data file
  const auto file_id =
      H5Fopen(FLAGS_testdata.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset x and y (x is input, y is output)
  const auto x_id = H5Dopen2(file_id, "/x", H5P_DEFAULT); // 'x' is the name of the 'dataset' in the testfile
  const auto y_id = H5Dopen2(file_id, "/y", H5P_DEFAULT); // 'y' is the name of the 'dataset' in the testfile

  // Get the dataset x dimensions
  const auto xspace = H5Dget_space(x_id);
  const auto xndims = H5Sget_simple_extent_ndims(xspace);
  assert(xndims == 4);

  hsize_t input_dims[xndims];
  H5Sget_simple_extent_dims(xspace, input_dims, NULL);
  if (input_dims[0] != FLAGS_batch_size) {
    std::cout << "data size does not match batch size specified!\n";
    return 1; // return error
  }
  std::cout << "input dimensions = " << input_dims[0] << " x " << input_dims[1]
            << " x " << input_dims[2] << " x " << input_dims[3] << "\n";

  // Read the dataset x and y
  check_success(
      H5Dread(x_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, x));
  check_success(
      H5Dread(y_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, y));

  // Close the dataset x and y
  check_success(H5Dclose(x_id));
  check_success(H5Dclose(y_id));

  // Close the file
  check_success(H5Fclose(file_id));

  // return success
  return 0;
}

static void loadModel(float *conv1, float *conv2, float *fc1, float *fc2) {
  // Open the model file
  const auto file_id = H5Fopen(FLAGS_model.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset
  const auto conv1_id = H5Dopen2(file_id, "/conv1", H5P_DEFAULT); // loaded from the model.hdf5 file
  const auto conv2_id = H5Dopen2(file_id, "/conv2", H5P_DEFAULT);
  const auto fc1_id   = H5Dopen2(file_id, "/fc1", H5P_DEFAULT);
  const auto fc2_id   = H5Dopen2(file_id, "/fc2", H5P_DEFAULT);

  // Read the dataset
  check_success(H5Dread(conv1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv1));
  check_success(H5Dread(conv2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv2));
  check_success(
      H5Dread(fc1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc1));
  check_success(
      H5Dread(fc2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc2));

  // Close the dataset x and y
  check_success(H5Dclose(conv1_id));
  check_success(H5Dclose(conv2_id));
  check_success(H5Dclose(fc1_id));
  check_success(H5Dclose(fc2_id));

  // Close the file
  check_success(H5Fclose(file_id));
}



// From book chapter Figure 16.4
// X in the input tensor
// W is the tensor of masks
// wdims is either conv1dims or conv2dims
// Y is the output after the convolution, ydims is the dimensions of the output tensor
static void conv_forward_valid(const float *X, const int xdims[4],
                               const float *W, const int wdims[4], float *Y,
                               const int ydims[4]) 
{
  const int filter_h   = wdims[0];
  const int filter_w   = wdims[1];
  const int C = wdims[2];
  const int M = ydims[3];
  auto getWIdx = [wdims] (int p, int q, int c, int m) {
      return p * wdims[1] * wdims[2] * wdims[3] +
             q * wdims[2] * wdims[3] + c * wdims[3] + m;};
  auto getXIdx = [xdims] (int i, int y, int x, int z) {
      return i * xdims[1] * xdims[2] * xdims[3] + y * xdims[2] * xdims[3] + x * xdims[3] + z;
  };
  auto getYIdx = [ydims] (int i, int row, int col, int num_feature_map) {
    return ((i * ydims[1] + row) * ydims[2] + col) * ydims[3] + num_feature_map;
  };
  // M output feature maps, C input feature maps
  // M*C masks
  // Y_i[:,:,m] = sum (Convolve2D X_i[:,:,c] and W[:,:,c,m])
  // Y_i[:,:,:] = sum (Convolve3D m copies of X_i[:,:,c] and W[:,:,c,:]) 
  for (const int i : range(0, ydims[0])) { // number of images
    for (const int m : range(0, M)) { // for each output feature map
      for (const int w : range(0, ydims[2])) { // for each output element
        for (const int h : range(0, ydims[1])) {   
          for (const int p : range(0, filter_h)) { // apply filter
            for (const int q : range(0, filter_w)) {
              for (const int c : range(0, C)) {  // for all input feature maps
                Y[getYIdx(i,h,w,m)] += X[getXIdx(i, h+p, w+q, c)] * W[getWIdx(p,q,c,m)];
              }
            }
          }
        }
      }
    }
  }
}

int multiplyArr(const int* arr, int n) {
  int prod = 1;
  
  for (int i = 0; i < n; i++) {
    prod *= arr[i];
  }
  return prod;
}

void easyConvWrapper(const float *X, const int xdims[4],
                     const float *W, const int wdims[4], float *Y,
                     const int ydims[4]) 
{
  const int W_out = ydims[2];
  const int H_out = ydims[1];
  const int W_grid = ceil(float(W_out)/float(TILE_SIZE)); // number of horizontal tiles per output map
  const int H_grid = ceil(float(H_out)/float(TILE_SIZE)); // number of vertical tiles per output map

  const int N = ydims[0]; //num images
  const int M = wdims[3]; //num output feature_maps

  const int Z = H_grid * W_grid; // total number of tiles
  
  const dim3 blockDim(TILE_SIZE, TILE_SIZE, 1);
  const dim3 gridDim(N, M, Z);
  
  int * deviceW_Grid;
  float* deviceX;
  float* deviceY;
  float* deviceW;
  int* deviceXDims;
  int* deviceYDims;
  int* deviceWDims;
  
  int sizeX = multiplyArr(xdims, 4)*sizeof(float);
  int sizeY = multiplyArr(ydims, 4)*sizeof(float);
  int sizeW = multiplyArr(wdims, 4)*sizeof(float);
  printf("sizeX = %d, sizeY = %d, sizeW = %d\n", sizeX, sizeY, sizeW);
  wbCheck(hipMalloc(&deviceX, sizeX));
  wbCheck(hipMalloc(&deviceY, sizeY));
  wbCheck(hipMalloc(&deviceW, sizeW));
  wbCheck(hipMalloc(&deviceXDims, 4 * sizeof(int)));
  wbCheck(hipMalloc(&deviceYDims, 4 * sizeof(int)));
  wbCheck(hipMalloc(&deviceWDims, 4 * sizeof(int)));
  wbCheck(hipMalloc(&deviceW_Grid, sizeof(int)));
  
  wbCheck(hipMemcpy(deviceX, X, sizeX, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceY, Y, sizeY, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceW, W, sizeW, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceXDims, xdims, 4 * sizeof(int), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceYDims, ydims, 4 * sizeof(int), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceWDims, wdims, 4 * sizeof(int), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceW_Grid, &W_grid, sizeof(int), hipMemcpyHostToDevice));
  
  easyConv<<<gridDim, blockDim>>>(deviceX, deviceXDims, deviceW, deviceWDims, deviceY, deviceYDims, deviceW_Grid);

  wbCheck(hipMemcpy(Y, deviceY, sizeY, hipMemcpyDeviceToHost));
  // for(int i = 0; i < multiplyArr(ydims,4); i++) {
  //   if (Y[i] > 0)
  //   printf("Y[%d] = %f\n", i,Y[i]);
  // }
  //Free CUDA Memory
}
//Y is output, X is input, W is the convolution mask
//XYZ Dims: Dimensions -- width, height, depth
__global__ void easyConv (const float *X, const int xdims[4],
                               const float *W, const int wdims[4], float *Y,
                               const int ydims[4], int* W_grid1)
{
  const int filter_h  = wdims[0];
  const int filter_w = wdims[1];
  const int C = wdims[2]; //num input feature_maps
  int W_grid = *W_grid1; // num tiles in horizontal direction
  auto getWIdx = [wdims] (int p, int q, int c, int m) {
      return p * wdims[1] * wdims[2] * wdims[3] +
             q * wdims[2] * wdims[3] + c * wdims[3] + m;};
  auto getXIdx = [xdims] (int i, int y, int x, int z) {
      return i * xdims[1] * xdims[2] * xdims[3] + y * xdims[2] * xdims[3] + x * xdims[3] + z;
  };
  auto getYIdx = [ydims] (int i, int row, int col, int num_feature_map) {
    return ((i * ydims[1] + row) * ydims[2] + col) * ydims[3] + num_feature_map;
  };
  int n, m, h, w, c, p, q;
  n = blockIdx.x;
  m = blockIdx.y;
  h = (blockIdx.z / W_grid) * TILE_SIZE + threadIdx.y;
  w = (blockIdx.z % W_grid) * TILE_SIZE + threadIdx.x;
  float acc = 0.0f;
  if (h < ydims[1] && w < ydims[2]) {
    for (p = 0; p < filter_h; p++){ // loop over KxK  filter
      for (q = 0; q < filter_w; q++){  
        for (c = 0;  c < C; c++) { // sum over all input feature maps      
            if (h+p < xdims[1] && w+q < xdims[2]) {
              acc += (X[getXIdx(n, h + p, w + q, c)] * W[getWIdx(p, q, c, m)]);
              //printf("X[%d,%d,%d,%d] = %f, W[%d,%d,%d,%d] = %f\n", n,h+p,w+q,c, X[getXIdx(n, h + p, w + q, c)], p,q,c,m, W[getWIdx(p, q, c, m)]);
            } 
        }
      }
    }
    Y[getYIdx(n, h, w, m)] = acc;
  }
  //printf("n = %d, h = %d, w = %d, m = %d, Y[%d,%d,%d,%d] = %f\n", n,h,w,m,n,h,w,m,Y[getYIdx(n, h, w, m)]);
  
}

// Recified linear unit 4d
static void relu4(float *X, const int xdims[4]) {
  for (const auto i : range(0, xdims[0] * xdims[1] * xdims[2] * xdims[3])) {
    X[i] = (X[i] < 0) ? 0 : X[i];
  }
}

// Recified linear unit 2d
static void relu2(float *X, const int xdims[2]) {
  for (const auto i : range(0, xdims[0] * xdims[1])) {
    X[i] = (X[i] < 0) ? 0 : X[i];
  }
}

// From book chapter Figure 16.5
static void average_pool(const float *X, const int xdims[4],
                         const int pool_size, float *Y, const int ydims[4]) 
{
  for (const auto i : range(0, ydims[0])) {
    for (const auto m : range(0, ydims[3])) {
      for (const auto w : range(0, ydims[2])) {
        for (const auto h : range(0, ydims[1])) {
          for (const auto p : range(0, pool_size)) {
            for (const auto q : range(0, pool_size)) {
              const auto yoffset =
                  ((i * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;
              const auto xoffset = i * xdims[1] * xdims[2] * xdims[3] +
                                   (pool_size * h + p) * xdims[2] * xdims[3] +
                                   (pool_size * w + q) * xdims[3] + m;
              Y[yoffset] += X[xoffset] / (1.0f * pool_size * pool_size);
            }
          }
        }
      }
    }
  }
}

static void fully_forward(const float *X, const int xdims[2], float *W,
                          const int wdims[2], float *Y, const int ydims[2]) 
{
  for (const auto i : range(0, xdims[0])) {
    for (const auto j : range(0, wdims[1])) {
      float sum = 0;
      for (const auto k : range(0, xdims[1])) {
        sum += X[i * xdims[1] + k] * W[k * wdims[1] + j];
      }
      Y[i * wdims[1] + j] = sum;
    }
  }
}

// Choose the guess with largest score
static void argmax(const float *X, const int xdims[2], int *Y) 
{
  for (const auto i : range(0, xdims[0])) {
    auto max_idx = 0;
    auto max     = X[i * xdims[1]];
    for (const auto j : range(0, xdims[1])) {
      const auto elem = X[(i * xdims[1]) + j];
      if (elem > max) {
        max_idx = j;
        max     = elem;
      }
    }
    Y[i] = max_idx;
  }
}


// Y_i[:,:,m] = sum (Convolve2D X_i[:,:,c] and W[:,:,c,m])
// blockIdx.x corresponds to m
// blockIdx.y corresponds to input_feature_map
// blockIdx.z corresponds to tiles
// threadIdx.x corresponds to m, threadIdx.y and threadIdx.z correspond col,row respectively 
// so each therad computes 2 partial convolutions
    // within a thread block c is the same.. so a thread block uses a patch of TILE_SIZE*2 of X_{i,c} and computes 
    // blockDim.x convolutions with different masks
    // a block reuses each element in a tile TILE_SIZE * TILE_SIZE * blockDim.x times
// we send all the convolutions to the kernel for one image
// as the result from an image comes back we sum the results on the CPU
void convolveWrapper(const float *X, const int xdims[4],
                               const float *W, const int wdims[4], float *Y,
                               const int ydims[4], bool useConstMemory) {
  const int num_images = xdims[0];
  const int C = wdims[2];
  const int M = wdims[3]; //num output feature_maps

  //const int batch_size = 1000; // going to do this many kernel calls at once
  
  const int W_out = ydims[2];
  const int H_out = ydims[1];
  const int W_grid = ceil(float(W_out)/float(TILE_SIZE)); // number of horizontal tiles per output map
  const int H_grid = ceil(float(H_out)/float(TILE_SIZE)); // number of vertical tiles per output map
  const int Z = H_grid * W_grid; // total number of tiles  
  
  int sizeX = multiplyArr(xdims, 4) * sizeof(float);
  int sizeY = multiplyArr(ydims, 4) * sizeof(float); // for each output_feature map element, all the c different values
                                                               // are stored in different locations, and the CPU will sum them
                                                              // we'll store them like Y[i,y,x,c,m] in device, and Y[y,x,c,m] in a temp arr on host
  int sizeW = multiplyArr(wdims, 4) * sizeof(float);  

  float* deviceX;
  float* deviceY;
  float* deviceW;
    
  int* deviceXDims;
  int* deviceYDims;
  int* deviceWDims; 

  // allocate memory 
  wbCheck(hipMalloc(&deviceX, sizeX));
  wbCheck(hipMalloc(&deviceY, sizeY));
  wbCheck(hipMalloc(&deviceXDims, 4 * sizeof(int)));
  wbCheck(hipMalloc(&deviceYDims, 4 * sizeof(int)));
  

 
  // copy memory
  wbCheck(hipMemcpy(deviceX, X, sizeX, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceY, Y, sizeY, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceXDims, xdims, 4 * sizeof(int), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceYDims, ydims, 4 * sizeof(int), hipMemcpyHostToDevice));

  dim3 gridDim((M/MperBlock) * num_images, C, Z); // ASSUMES - that M is a multiple of 16
  dim3 blockDim(MperBlock,TILE_SIZE,TILE_SIZE);
  

  if (useConstMemory) {  
    wbCheck(hipMemcpyToSymbol(HIP_SYMBOL(mask1), W, sizeW));
    convolve1<<<gridDim, blockDim>>>(deviceX, deviceXDims, deviceY, deviceYDims, W_grid, num_images);
  }
  else {
    wbCheck(hipMalloc(&deviceWDims, 4 * sizeof(int)));
    wbCheck(hipMalloc(&deviceW, sizeW));
    wbCheck(hipMemcpy(deviceWDims, wdims, 4 * sizeof(int), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(deviceW, W, sizeW, hipMemcpyHostToDevice));
    convolve<<<gridDim, blockDim>>>(deviceX, deviceXDims, deviceW, deviceWDims, deviceY, deviceYDims, W_grid, num_images);
  }
 

  hipMemcpy(Y, deviceY, sizeY, hipMemcpyDeviceToHost);

  wbCheck(hipFree(deviceX));
  wbCheck(hipFree(deviceY));
  if(!useConstMemory) wbCheck(hipFree(deviceW));
  wbCheck(hipFree(deviceXDims));
  wbCheck(hipFree(deviceYDims));
  wbCheck(hipFree(deviceWDims));
   
}


//change so each thread computes two convolutions
__global__ void convolve(const float *X, const int xdims[4],
                        const float *W, const int wdims[4], float *Y,
                        const int ydims[4], int W_grid, int num_images) {
  
  
  #define tx threadIdx.x
  #define ty threadIdx.y
  #define tz threadIdx.z
  
  const int m = (blockIdx.x / num_images) * blockDim.x + tx;
  const int h = (blockIdx.z / W_grid) * TILE_SIZE + tz;
  const int w = (blockIdx.z % W_grid) * TILE_SIZE + ty;
  const int n = blockIdx.x % num_images;
  const int c = blockIdx.y; // each thread does a convolution of X[n, h:h+5, w:w+5,m] with W[:, :, c, m]

  const int input_TILE_SIZE = TILE_SIZE + FILTER_SIZE - 1; // ASSUMES :  TILE_SIZE + FILTER_SIZE - 1 < 2*TILE_SIZE 
  
  __shared__ float sharedX[input_TILE_SIZE][input_TILE_SIZE];
  __shared__ float sharedW[FILTER_SIZE][FILTER_SIZE][MperBlock];

  if (tx == 0) {
    
      int hbase = (h-tz);
      int wbase = w-ty;
      for(int i = h; i < hbase + input_TILE_SIZE; i+= TILE_SIZE) {
        for (int j = w; j < wbase + input_TILE_SIZE; j+= TILE_SIZE) {
          if (i < xdims[1] && j < xdims[2])
            sharedX[i-hbase][j-wbase] = X[n * xdims[1] * xdims[2] * xdims[3] + i * xdims[2] * xdims[3] + j * xdims[3] + c];
          else
            sharedX[i-hbase][j-wbase] = 0.0f;
        }
      }
  }
  if (tz < FILTER_SIZE && ty < FILTER_SIZE) {
    sharedW[tz][ty][tx] = W[tz * wdims[1] * wdims[2] * wdims[3] + ty * wdims[2] * wdims[3] + c * wdims[3] + m]; // ASSUMES : blockdim.z, blockdim.y >= filter_h,filter_w
  }
  
  __syncthreads();
  


  if (h < ydims[1] && w < ydims[2]) {

    float sum = 0.0f;

    for (int p = 0; p < FILTER_SIZE; p++) {
      for (int q = 0; q < FILTER_SIZE; q++) {
        sum += sharedX[tz + p][ty + q] * sharedW[p][q][tx];
      }
    }
    atomicAdd(&(Y[n * ydims[1] * ydims[2] * ydims[3] + h * ydims[2] * ydims[3] + w * ydims[3] + m]), sum);

  }

}

__global__ void convolve1(const float *X, const int xdims[4], float *Y,
                          const int ydims[4], int W_grid, int num_images) {
  
  
  #define tx threadIdx.x
  #define ty threadIdx.y
  #define tz threadIdx.z
  
  const int m = (blockIdx.x / num_images) * blockDim.x + tx;
  const int h = (blockIdx.z / W_grid) * TILE_SIZE + tz;
  const int w = (blockIdx.z % W_grid) * TILE_SIZE + ty;
  const int n = blockIdx.x % num_images;
  const int c = blockIdx.y; // each thread does a convolution of X[n, h:h+5, w:w+5,m] with W[:, :, c, m]

  const int input_TILE_SIZE = TILE_SIZE + FILTER_SIZE - 1; // ASSUMES :  TILE_SIZE + FILTER_SIZE - 1 < 2*TILE_SIZE 
  
  __shared__ float sharedX[input_TILE_SIZE][input_TILE_SIZE];

  if (tx == 0) {
    
      int hbase = (h-tz);
      int wbase = w-ty;
      for(int i = h; i < hbase + input_TILE_SIZE; i+= TILE_SIZE) {
        for (int j = w; j < wbase + input_TILE_SIZE; j+= TILE_SIZE) {
          if (i < xdims[1] && j < xdims[2])
            sharedX[i-hbase][j-wbase] = X[n * xdims[1] * xdims[2] * xdims[3] + i * xdims[2] * xdims[3] + j * xdims[3] + c];
          else
            sharedX[i-hbase][j-wbase] = 0.0f;
        }
      }
  }
  
  __syncthreads();
  

  if (h < ydims[1] && w < ydims[2]) {

    float sum = 0.0f;

    for (int p = 0; p < FILTER_SIZE; p++) {
      for (int q = 0; q < FILTER_SIZE; q++) {
        sum += sharedX[tz + p][ty + q] * mask1[p][q][m];
      }
    }
    atomicAdd(&(Y[n * ydims[1] * ydims[2] * ydims[3] + h * ydims[2] * ydims[3] + w * ydims[3] + m]), sum);

  }

}


// Forward operation for the CNN, a combination of conv layer + average pooling
// + relu
void forward_operation(float *x, float *conv1, float *conv2, float *fc1,
                       float *fc2, int *out) {
  // conv layer
  const int adims[] = {xdims[0], (xdims[1] - conv1dims[0] + 1),
                       (xdims[2] - conv1dims[1] + 1), conv1dims[3]};
  auto a = zeros<float>(adims);
  convolveWrapper(x, xdims, conv1, conv1dims, a, adims, true);
  //easyConvWrapper(x, xdims, conv1, conv1dims, a, adims);
  //conv_forward_valid(x, xdims, conv1, conv1dims, a, adims);
  
  /// relu layer
  relu4(a, adims);

  // average pooling
  const int pool_size = 2;
  const int bdims[]   = {adims[0], adims[1] / pool_size, adims[2] / pool_size,
                       adims[3]};
  auto b = zeros<float>(bdims);
  average_pool(a, adims, pool_size, b, bdims);

  // conv layer
  const int cdims[] = {bdims[0], (bdims[1] - conv2dims[0] + 1),
                       (bdims[2] - conv2dims[1] + 1), conv2dims[3]};
  auto c = zeros<float>(cdims);
  convolveWrapper(b, bdims, conv2, conv2dims, c, cdims, false);
  //easyConvWrapper(b, bdims, conv2, conv2dims, c, cdims);
  //conv_forward_valid(b, bdims, conv2, conv2dims, c, cdims);
  // relu
  relu4(c, cdims);

  // average pooling
  const int ddims[] = {cdims[0], cdims[1] / pool_size, cdims[2] / pool_size,
                       cdims[3]};
  auto d = zeros<float>(ddims);
  average_pool(c, cdims, pool_size, d, ddims);

  // reshape
  const int ddims2[] = {ddims[0], ddims[1] * ddims[2] * ddims[3]};

  // matrix multiplication
  const int edims[] = {ddims[0], fc1dims[1]};
  auto e            = zeros<float>(edims);
  fully_forward(d, ddims2, fc1, fc1dims, e, edims);

  // relu
  relu2(e, edims);

  // matrix multiplication
  const int fdims[] = {edims[0], fc2dims[1]};
  auto f            = zeros<float>(fdims);
  fully_forward(e, edims, fc2, fc2dims, f, fdims);

  argmax(f, fdims, out);

  delete[] a;
  delete[] b;
  delete[] c;
  delete[] d;
  delete[] e;
  delete[] f;
}



int main(int argc, char **argv) {

  if (argc != 3 && argc != 4) {
    std::cerr << "\n"
              << "This program performs the forward opertion step for "
                 "Convolutional Neural Network(CNN).  "
                 "Sample usage: \n"
              << argv[0]
              << " [../data/test10.hdf5] [../data/model.hdf5] [10]\n";
    return -1;
  }
  FLAGS_testdata = std::string(argv[1]);
  FLAGS_model    = std::string(argv[2]);
  if (argc == 3) {
    const std::map<std::string, int> default_batch_sizes{
        {"../data/test2.hdf5", 2},
        {"../data/test10.hdf5", 10},
        {"../data/test100.hdf5", 100},
        {"../data/testfull.hdf5", 10000}};
    const auto batch_size_in_map = default_batch_sizes.find(FLAGS_testdata);
    if (batch_size_in_map == default_batch_sizes.end()) {
      std::cerr << "\nERROR:: Unrecognized file " << FLAGS_testdata << " batch_size must be specified.\n";
      return -1;
    }
    FLAGS_batch_size = batch_size_in_map->second;
  } else if (argc == 4) {
    FLAGS_batch_size = atoi(argv[3]);
  }
  xdims[0] = FLAGS_batch_size; // number of images
  rdims[0] = FLAGS_batch_size; // number of images

  // Load data into x and y
  float *x = allocate<float>(xdims);
  float *y = allocate<float>(rdims);
  loadData(x, y);

  // Load model
  float *conv1 = allocate<float>(conv1dims);
  float *conv2 = allocate<float>(conv2dims);
  float *fc1   = allocate<float>(fc1dims);
  float *fc2   = allocate<float>(fc2dims);
  loadModel(conv1, conv2, fc1, fc2);

  // Perform foward opertion
  int *out = zeros<int>(FLAGS_batch_size);

  // get start time
  const auto start = now();

  forward_operation(x, conv1, conv2, fc1, fc2, out);

  // get end time
  const auto end = now();

  // get elapsed time in milliseconds
  const auto elapsed =
      std::chrono::duration<double, std::milli>(end - start).count();

  // Get reference
  int *ref = zeros<int>(FLAGS_batch_size);
  argmax(y, rdims, ref);

  // Calculate correctness
  int num_correct = 0;
  for (const auto i : range(0, FLAGS_batch_size)) {
    if (out[i] == ref[i]) {
      num_correct++;
    }
  }
  std::cout << "Done with " << FLAGS_batch_size << " queries in "
            << "elapsed = " << elapsed << " milliseconds. Correctness: "
            << static_cast<float>(num_correct) / FLAGS_batch_size << "\n";

  delete[] x;
  delete[] y;
  delete[] conv1;
  delete[] conv2;
  delete[] fc1;
  delete[] fc2;
  delete[] out;
  delete[] ref;

  return 0;
}
