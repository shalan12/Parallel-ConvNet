#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstddef>
#include <iostream>
#include <numeric>
#include <map>
#include <sys/time.h>
#include <valarray>

#include <hdf5.h>

#include "range.hpp"
#include "utils.hpp"

#define NUM_ROWS 28
#define NUM_COLS 28
#define NUM_CHANNELS 1
#define NUM_DIGITS 10
#define TILE_SIZE 16

#define POOL_TILE_SIZE 4
#define POOL_BLOCK_OUTPUT_FEATURE 32 //so all accesses in a warp are consecutive

#define RELU_TILE_SIZE 10

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      printf("Failed to run stmt %s\n", #stmt);                         \
      printf("ERROR - Got CUDA error ...  %s\n", hipGetErrorString(err));      \
      return ;                                                            \
    }                                                                     \
  } while (0)

static int FLAGS_batch_size = 10000; // number of images....actual value changes at runtime
static std::string FLAGS_testdata{};
static std::string FLAGS_model{};

// Data and reference data dimensions
static int xdims[] = {FLAGS_batch_size, NUM_ROWS, NUM_COLS, NUM_CHANNELS};
static int rdims[] = {FLAGS_batch_size, NUM_DIGITS};

// Model dimensions
static int conv1dims[] = {5, 5, 1, 32}; // rows, cols, #input_feature maps, #output_feature_maps
static int conv2dims[] = {5, 5, 32, 64}; // rows, cols, #input_feature maps, #output_feature_maps
static int fc1dims[]   = {1024, 128}; // not important for convolution or subsampling layers
static int fc2dims[]   = {128, 10}; // not important for convolution or subsampling layers

void easyConvWrapper (const float *X, const int xdims[4], const float *W, const int wdims[4], float *Y, const int ydims[4]);
__global__ void easyConv (const float *X, const int xdims[4],
                               const float *W, const int wdims[4], float *Y,
                               const int ydims[4], int* W_grid);
void parallel_pool_wrapper(const float *X, const int xdims[4],
                         const int pool_size, float *Y, const int ydims[4]);
__global__ void parallel_pool(const float *X, const int xdims[4],
                         const int pool_size, float *Y, const int ydims[4], int wGrid);
void parallelRelu4Wrapper(float *X, const int xdims[4]);
__global__ void parallelRelu4(float *X, const int size);

void parallelFullyForwardWrapper(const float *X, const int xdims[2], float *W,
                          const int wdims[2], float *Y, const int ydims[2]);

__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns);

static int loadData(float *x, float *y) {
  // Open the data file
  const auto file_id =
      H5Fopen(FLAGS_testdata.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset x and y (x is input, y is output)
  const auto x_id = H5Dopen2(file_id, "/x", H5P_DEFAULT); // 'x' is the name of the 'dataset' in the testfile
  const auto y_id = H5Dopen2(file_id, "/y", H5P_DEFAULT); // 'y' is the name of the 'dataset' in the testfile

  // Get the dataset x dimensions
  const auto xspace = H5Dget_space(x_id);
  const auto xndims = H5Sget_simple_extent_ndims(xspace);
  assert(xndims == 4);

  hsize_t input_dims[xndims];
  H5Sget_simple_extent_dims(xspace, input_dims, NULL);
  if (input_dims[0] != FLAGS_batch_size) {
    std::cout << "data size does not match batch size specified!\n";
    return 1; // return error
  }
  std::cout << "input dimensions = " << input_dims[0] << " x " << input_dims[1]
            << " x " << input_dims[2] << " x " << input_dims[3] << "\n";

  // Read the dataset x and y
  check_success(
      H5Dread(x_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, x));
  check_success(
      H5Dread(y_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, y));

  // Close the dataset x and y
  check_success(H5Dclose(x_id));
  check_success(H5Dclose(y_id));

  // Close the file
  check_success(H5Fclose(file_id));

  // return success
  return 0;
}

static void loadModel(float *conv1, float *conv2, float *fc1, float *fc2) {
  // Open the model file
  const auto file_id = H5Fopen(FLAGS_model.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

  // Open the dataset
  const auto conv1_id = H5Dopen2(file_id, "/conv1", H5P_DEFAULT); // loaded from the model.hdf5 file
  const auto conv2_id = H5Dopen2(file_id, "/conv2", H5P_DEFAULT);
  const auto fc1_id   = H5Dopen2(file_id, "/fc1", H5P_DEFAULT);
  const auto fc2_id   = H5Dopen2(file_id, "/fc2", H5P_DEFAULT);

  // Read the dataset
  check_success(H5Dread(conv1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv1));
  check_success(H5Dread(conv2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
                        H5P_DEFAULT, conv2));
  check_success(
      H5Dread(fc1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc1));
  check_success(
      H5Dread(fc2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc2));

  // Close the dataset x and y
  check_success(H5Dclose(conv1_id));
  check_success(H5Dclose(conv2_id));
  check_success(H5Dclose(fc1_id));
  check_success(H5Dclose(fc2_id));

  // Close the file
  check_success(H5Fclose(file_id));
}



// From book chapter Figure 16.4
// X in the input tensor
// W is the tensor of masks
// wdims is either conv1dims or conv2dims
// Y is the output after the convolution, ydims is the dimensions of the output tensor
static void conv_forward_valid(const float *X, const int xdims[4],
                               const float *W, const int wdims[4], float *Y,
                               const int ydims[4]) 
{
  const int filter_h   = wdims[0];
  const int filter_w   = wdims[1];
  const int C = wdims[2];
  const int M = ydims[3];
  auto getWIdx = [wdims] (int p, int q, int c, int m) {
      return p * wdims[1] * wdims[2] * wdims[3] +
             q * wdims[2] * wdims[3] + c * wdims[3] + m;};
  auto getXIdx = [xdims] (int i, int y, int x, int z) {
      return i * xdims[1] * xdims[2] * xdims[3] + y * xdims[2] * xdims[3] + x * xdims[3] + z;
  };
  auto getYIdx = [ydims] (int i, int row, int col, int num_feature_map) {
    return ((i * ydims[1] + row) * ydims[2] + col) * ydims[3] + num_feature_map;
  };
  // M output feature maps, C input feature maps
  // M*C masks
  // Y_i[:,:,m] = sum (Convolve2D X_i[:,:,c] and W[:,:,c,m])
  // Y_i[:,:,:] = sum (Convolve3D m copies of X_i[:,:,c] and W[:,:,c,:]) 
  for (const int i : range(0, ydims[0])) { // number of images
    for (const int m : range(0, M)) { // for each output feature map
      for (const int w : range(0, ydims[2])) { // for each output element
        for (const int h : range(0, ydims[1])) {   
          for (const int p : range(0, filter_h)) { // apply filter
            for (const int q : range(0, filter_w)) {
              for (const int c : range(0, C)) {  // for all input feature maps
                Y[getYIdx(i,h,w,m)] += X[getXIdx(i, h+p, w+q, c)] * W[getWIdx(p,q,c,m)];
              }
            }
          }
        }
      }
    }
  }
}

int multiplyArr(const int* arr, int n) {
  int prod = 1;
  
  for (int i = 0; i < n; i++) {
    prod *= arr[i];
  }
  return prod;
}

void easyConvWrapper(const float *X, const int xdims[4],
                     const float *W, const int wdims[4], float *Y,
                     const int ydims[4]) {
  const int W_out = ydims[2];
  const int H_out = ydims[1];
  const int W_grid = ceil(float(W_out)/float(TILE_SIZE)); // number of horizontal tiles per output map
  const int H_grid = ceil(float(H_out)/float(TILE_SIZE)); // number of vertical tiles per output map

  const int N = ydims[0]; //num images
  const int M = wdims[3]; //num output feature_maps

  const int Z = H_grid * W_grid; // total number of tiles
  
  const dim3 blockDim(TILE_SIZE, TILE_SIZE, 1);
  const dim3 gridDim(N, M, Z);
  
  int * deviceW_Grid;
  float* deviceX;
  float* deviceY;
  float* deviceW;
  int* deviceXDims;
  int* deviceYDims;
  int* deviceWDims;
  
  int sizeX = multiplyArr(xdims, 4)*sizeof(float);
  int sizeY = multiplyArr(ydims, 4)*sizeof(float);
  int sizeW = multiplyArr(wdims, 4)*sizeof(float);
  
  wbCheck(hipMalloc(&deviceX, sizeX));
  wbCheck(hipMalloc(&deviceY, sizeY));
  wbCheck(hipMalloc(&deviceW, sizeW));
  wbCheck(hipMalloc(&deviceXDims, 4 * sizeof(int)));
  wbCheck(hipMalloc(&deviceYDims, 4 * sizeof(int)));
  wbCheck(hipMalloc(&deviceWDims, 4 * sizeof(int)));
  wbCheck(hipMalloc(&deviceW_Grid, sizeof(int)));
  
  wbCheck(hipMemcpy(deviceX, X, sizeX, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceY, Y, sizeY, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceW, W, sizeW, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceXDims, xdims, 4 * sizeof(int), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceYDims, ydims, 4 * sizeof(int), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceWDims, wdims, 4 * sizeof(int), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceW_Grid, &W_grid, sizeof(int), hipMemcpyHostToDevice));
  
  easyConv<<<gridDim, blockDim>>>(deviceX, deviceXDims, deviceW, deviceWDims, deviceY, deviceYDims, deviceW_Grid);

  wbCheck(hipMemcpy(Y, deviceY, sizeY, hipMemcpyDeviceToHost));
  // for(int i = 0; i < multiplyArr(ydims,4); i++) {
  //   if (Y[i] > 0)
  //   printf("Y[%d] = %f\n", i,Y[i]);
  // }
  //Free CUDA Memory
}
//Y is output, X is input, W is the convolution mask
//XYZ Dims: Dimensions -- width, height, depth
__global__ void easyConv (const float *X, const int xdims[4],
                               const float *W, const int wdims[4], float *Y,
                               const int ydims[4], int* W_grid1){
  const int filter_h  = wdims[0];
  const int filter_w = wdims[1];
  const int C = wdims[2]; //num input feature_maps
  int W_grid = *W_grid1; // num tiles in horizontal direction
  auto getWIdx = [wdims] (int p, int q, int c, int m) {
      return p * wdims[1] * wdims[2] * wdims[3] +
             q * wdims[2] * wdims[3] + c * wdims[3] + m;};
  auto getXIdx = [xdims] (int i, int y, int x, int z) {
      return i * xdims[1] * xdims[2] * xdims[3] + y * xdims[2] * xdims[3] + x * xdims[3] + z;
  };
  auto getYIdx = [ydims] (int i, int row, int col, int num_feature_map) {
    return ((i * ydims[1] + row) * ydims[2] + col) * ydims[3] + num_feature_map;
  };
  int n, m, h, w, c, p, q;
  n = blockIdx.x;
  m = blockIdx.y;
  h = (blockIdx.z / W_grid) * TILE_SIZE + threadIdx.y;
  w = (blockIdx.z % W_grid) * TILE_SIZE + threadIdx.x;
  float acc = 0.0f;
  if (h < ydims[1] && w < ydims[2]) {
    for (p = 0; p < filter_h; p++){ // loop over KxK  filter
      for (q = 0; q < filter_w; q++){  
        for (c = 0;  c < C; c++) { // sum over all input feature maps      
            if (h+p < xdims[1] && w+q < xdims[2]) {
              acc += (X[getXIdx(n, h + p, w + q, c)] * W[getWIdx(p, q, c, m)]);
              //printf("X[%d,%d,%d,%d] = %f, W[%d,%d,%d,%d] = %f\n", n,h+p,w+q,c, X[getXIdx(n, h + p, w + q, c)], p,q,c,m, W[getWIdx(p, q, c, m)]);
            } 
        }
      }
    }
    Y[getYIdx(n, h, w, m)] = acc;
  }
  //printf("n = %d, h = %d, w = %d, m = %d, Y[%d,%d,%d,%d] = %f\n", n,h,w,m,n,h,w,m,Y[getYIdx(n, h, w, m)]);
  
}

// Recified linear unit 4d
static void relu4(float *X, const int xdims[4]) {
  for (const auto i : range(0, xdims[0] * xdims[1] * xdims[2] * xdims[3])) {
    X[i] = (X[i] < 0) ? 0 : X[i];
  }
}

void parallelRelu4Wrapper(float *X, const int xdims[4]) {
  const int numElemsX  = multiplyArr(xdims, 4);
  int sizeX = numElemsX * sizeof(float);
  float * deviceX;
  wbCheck(hipMalloc(&deviceX, sizeX));
  wbCheck(hipMemcpy(deviceX, X, sizeX, hipMemcpyHostToDevice));
  const int numThreads = 512;
  const int numBlocks = numElemsX/(numThreads*RELU_TILE_SIZE);
  parallelRelu4<<<numBlocks, numThreads>>>(deviceX, numElemsX);
  wbCheck(hipMemcpy(X, deviceX, sizeX, hipMemcpyDeviceToHost));
}

__global__ void parallelRelu4(float *X, const int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = 0; i < RELU_TILE_SIZE; i++) {
    if (idx > size) break;
    else {
      X[idx] = (X[idx] < 0) ? 0 : X[idx];
      idx += stride;
    }
  } 
}

// Recified linear unit 2d
static void relu2(float *X, const int xdims[2]) {
  for (const auto i : range(0, xdims[0] * xdims[1])) {
    X[i] = (X[i] < 0) ? 0 : X[i];
  }
}

// From book chapter Figure 16.5
static void average_pool(const float *X, const int xdims[4],
                         const int pool_size, float *Y, const int ydims[4]) 
{
  for (const auto i : range(0, ydims[0])) {
    for (const auto m : range(0, ydims[3])) {
      for (const auto w : range(0, ydims[2])) {
        for (const auto h : range(0, ydims[1])) {
          for (const auto p : range(0, pool_size)) {
            for (const auto q : range(0, pool_size)) {
              const auto yoffset =
                  ((i * ydims[1] + h) * ydims[2] + w) * ydims[3] + m;
              const auto xoffset = i * xdims[1] * xdims[2] * xdims[3] +
                                   (pool_size * h + p) * xdims[2] * xdims[3] +
                                   (pool_size * w + q) * xdims[3] + m;
              Y[yoffset] += X[xoffset] / (1.0f * pool_size * pool_size);
            }
          }
        }
      }
    }
  }
}

void parallel_pool_wrapper(const float *X, const int xdims[4],
                         const int pool_size, float *Y, const int ydims[4]) {
  
  const int H_grid = ceil(float(ydims[1])/float(POOL_TILE_SIZE));
  const int W_grid = ceil(float(ydims[2])/float(POOL_TILE_SIZE));
  const int Z = H_grid * W_grid;
  
  dim3 blockDim(POOL_BLOCK_OUTPUT_FEATURE, POOL_TILE_SIZE, POOL_TILE_SIZE);
  dim3 gridDim(ydims[3]/POOL_BLOCK_OUTPUT_FEATURE, Z ,ydims[0]); // ASSUMES 32 | M
  
  float* deviceX;
  float* deviceY;
  int* deviceXDims;
  int* deviceYDims;

  int sizeX = multiplyArr(xdims, 4) * sizeof(float);
  int sizeY = multiplyArr(ydims, 4) * sizeof(float);
  
  wbCheck(hipMalloc(&deviceX, sizeX));
  wbCheck(hipMalloc(&deviceY, sizeY));
  wbCheck(hipMalloc(&deviceXDims, 4 * sizeof(int)));
  wbCheck(hipMalloc(&deviceYDims, 4 * sizeof(int)));
  
  wbCheck(hipMemcpy(deviceX, X, sizeX, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceY, Y, sizeY, hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceXDims, xdims, 4 * sizeof(int), hipMemcpyHostToDevice));
  wbCheck(hipMemcpy(deviceYDims, ydims, 4 * sizeof(int), hipMemcpyHostToDevice));


  parallel_pool<<<gridDim, blockDim>>>(deviceX, deviceXDims, pool_size, deviceY, deviceYDims, W_grid);

  wbCheck(hipMemcpy(Y, deviceY, sizeY, hipMemcpyDeviceToHost));
  

  wbCheck(hipFree(deviceX));
  wbCheck(hipFree(deviceY));
  wbCheck(hipFree(deviceXDims));
  wbCheck(hipFree(deviceYDims));
}

__global__ void parallel_pool(const float *X, const int xdims[4],
                         const int pool_size, float *Y, const int ydims[4], int wGrid) {
  int h_base = (blockIdx.y / wGrid) * POOL_TILE_SIZE;
  int w_base = (blockIdx.y % wGrid) * POOL_TILE_SIZE;
  int h0 = threadIdx.z;
  int w0 = threadIdx.y;
  int m = blockIdx.x * blockDim.x + threadIdx.x;
  int h = h_base + h0;
  int w = w_base + w0;
  int n = blockIdx.z;

  float sum = 0.0f;
  if(h < ydims[1] && w < ydims[2]){
    for(int i = 0; i < pool_size; i++){
      for(int j = 0; j < pool_size; j++){
        if (h + i < xdims[1] && w+j < xdims[2])
        sum += X[n * xdims[1] * xdims[2] * xdims[3] + (h*pool_size + i) * xdims[2] * xdims[3] + (w * pool_size+j) * xdims[3] + m]/(1.0f* pool_size* pool_size);
      }
    }
    Y[n * ydims[1] * ydims[2] * ydims[3] + h * ydims[2] * ydims[3] + w * ydims[3] + m] = sum;
  }
}

void parallelFullyForwardWrapper(const float *X, const int xdims[2], 
                                       float *W, const int wdims[2], 
                                       float *Y, const int ydims[2]){
  //Grid Dimensions
  dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
  dim3 dimGrid(ceil(float(ydims[1])/float(TILE_SIZE)), ceil(float(ydims[0])/float(TILE_SIZE)), 1);
  
  //Device Memory Allocation
  float* deviceX; 
  float* deviceW;
  float* deviceY;
  int sizeX, sizeY, sizeW;
  sizeX = xdims[0]*xdims[1]*sizeof(float);
  sizeY = ydims[0]*ydims[1]*sizeof(float);
  sizeW = wdims[0]*wdims[1]*sizeof(float);
  hipMalloc(&deviceX, sizeX);
  hipMalloc(&deviceW, sizeW);
  hipMalloc(&deviceY, sizeY);

  //Memcpy'ing
   wbCheck(hipMemcpy(deviceX, X, sizeX, hipMemcpyHostToDevice));
   wbCheck(hipMemcpy(deviceW, W, sizeW, hipMemcpyHostToDevice));
   wbCheck(hipMemcpy(deviceY, Y, sizeY, hipMemcpyHostToDevice));

  matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceX, deviceW, deviceY,
                                     xdims[0], xdims[1],
                                     wdims[0], wdims[1],
                                     ydims[0], ydims[1]);
  wbCheck(hipMemcpy(Y, deviceY, sizeY, hipMemcpyDeviceToHost));
  /*for(int i = 0; i < ydims[0]*ydims[1]; i++){
    printf("Y[%d] = %f\n",i, Y[i]);
  }*/
  //Frees
  wbCheck(hipFree(deviceX)); wbCheck(hipFree(deviceY)); wbCheck(hipFree(deviceW));
}

__device__ int divAndCeil(int n, int d) {
  int t = n/d;  
  return t + (n%d > 0);
}

__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //Shared Memory
  __shared__ float subTileA[TILE_SIZE][TILE_SIZE];
  __shared__ float subTileB[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
  //Row and Column Indices
    int Row = by * TILE_SIZE + ty;
    int Col = bx * TILE_SIZE + tx;
    float Pvalue = 0;
    
  //Cooperating to load each of the subtiles
    for (int m = 0; m < (numAColumns+TILE_SIZE-1/TILE_SIZE); m++) { //Could have also used numBRows (this is 'width') in the example
       if (Row < numARows && m*TILE_SIZE+tx < numAColumns) //If the index is in bounds for A
          subTileA[ty][tx] = A[Row*numAColumns + m*TILE_SIZE+tx];
       else
          subTileA[ty][tx] = 0.0;
       if (Col < numBColumns && m*TILE_SIZE+ty < numBRows) //If the index is in bounds for B
          subTileB[ty][tx] = B[(m*TILE_SIZE+ty)*numBColumns+Col];
       else
          subTileB[ty][tx] = 0.0;

       __syncthreads();
       for (int k = 0; k < TILE_SIZE; k++)
          Pvalue += subTileA[ty][k] * subTileB[k][tx];
       __syncthreads();
    }
    if (Row < numCRows && Col < numCColumns)
       C[Row*numCColumns+Col] = Pvalue;
 
}

static void fully_forward(const float *X, const int xdims[2], float *W,
                          const int wdims[2], float *Y, const int ydims[2]) 
{
  for (const auto i : range(0, xdims[0])) {
    for (const auto j : range(0, wdims[1])) {
      float sum = 0;
      for (const auto k : range(0, xdims[1])) {
        sum += X[i * xdims[1] + k] * W[k * wdims[1] + j];
      }
      Y[i * wdims[1] + j] = sum;
    }
  }
}

// Choose the guess with largest score
static void argmax(const float *X, const int xdims[2], int *Y) 
{
  for (const auto i : range(0, xdims[0])) {
    auto max_idx = 0;
    auto max     = X[i * xdims[1]];
    for (const auto j : range(0, xdims[1])) {
      const auto elem = X[(i * xdims[1]) + j];
      if (elem > max) {
        max_idx = j;
        max     = elem;
      }
    }
    Y[i] = max_idx;
  }
}

// Forward operation for the CNN, a combination of conv layer + average pooling
// + relu
void forward_operation(float *x, float *conv1, float *conv2, float *fc1,
                       float *fc2, int *out) {
  // conv layer
  const int adims[] = {xdims[0], (xdims[1] - conv1dims[0] + 1),
                       (xdims[2] - conv1dims[1] + 1), conv1dims[3]};
  auto a = zeros<float>(adims);
  easyConvWrapper(x, xdims, conv1, conv1dims, a, adims);
  //conv_forward_valid(x, xdims, conv1, conv1dims, a, adims);
  
  /// relu layer
  //relu4(a, adims);
  parallelRelu4Wrapper(a, adims);


  // average pooling
  const int pool_size = 2;
  const int bdims[]   = {adims[0], adims[1] / pool_size, adims[2] / pool_size,
                       adims[3]};
  auto b = zeros<float>(bdims);
  parallel_pool_wrapper(a, adims, pool_size, b, bdims);
  //average_pool(a, adims, pool_size, b, bdims);

  // conv layer
  const int cdims[] = {bdims[0], (bdims[1] - conv2dims[0] + 1),
                       (bdims[2] - conv2dims[1] + 1), conv2dims[3]};
  auto c = zeros<float>(cdims);
  easyConvWrapper(b, bdims, conv2, conv2dims, c, cdims);
  //conv_forward_valid(b, bdims, conv2, conv2dims, c, cdims);
  // relu
  //relu4(c, cdims);
  parallelRelu4Wrapper(c, cdims);
  
  // average pooling
  const int ddims[] = {cdims[0], cdims[1] / pool_size, cdims[2] / pool_size,
                       cdims[3]};
  auto d = zeros<float>(ddims);
  parallel_pool_wrapper(c, cdims, pool_size, d, ddims);
  //average_pool(c, cdims, pool_size, d, ddims);

  // reshape
  const int ddims2[] = {ddims[0], ddims[1] * ddims[2] * ddims[3]};

  // matrix multiplication
  const int edims[] = {ddims[0], fc1dims[1]};
  auto e            = zeros<float>(edims);
  //fully_forward(d, ddims2, fc1, fc1dims, e, edims);
  parallelFullyForwardWrapper(d, ddims2, fc1, fc1dims, e, edims);

  // relu
  relu2(e, edims);

  // matrix multiplication
  const int fdims[] = {edims[0], fc2dims[1]};

  auto f            = zeros<float>(fdims);
  //fully_forward(e, edims, fc2, fc2dims, f, fdims);
  parallelFullyForwardWrapper(e, edims, fc2, fc2dims, f, fdims);

  argmax(f, fdims, out);

  delete[] a;
  delete[] b;
  delete[] c;
  delete[] d;
  delete[] e;
  delete[] f;
}



int main(int argc, char **argv) {

  if (argc != 3 && argc != 4) {
    std::cerr << "\n"
              << "This program performs the forward opertion step for "
                 "Convolutional Neural Network(CNN).  "
                 "Sample usage: \n"
              << argv[0]
              << " [../data/test10.hdf5] [../data/model.hdf5] [10]\n";
    return -1;
  }
  FLAGS_testdata = std::string(argv[1]);
  FLAGS_model    = std::string(argv[2]);
  if (argc == 3) {
    const std::map<std::string, int> default_batch_sizes{
        {"../data/test2.hdf5", 2},
        {"../data/test10.hdf5", 10},
        {"../data/test100.hdf5", 100},
        {"../data/testfull.hdf5", 10000}};
    const auto batch_size_in_map = default_batch_sizes.find(FLAGS_testdata);
    if (batch_size_in_map == default_batch_sizes.end()) {
      std::cerr << "\nERROR:: Unrecognized file " << FLAGS_testdata << " batch_size must be specified.\n";
      return -1;
    }
    FLAGS_batch_size = batch_size_in_map->second;
  } else if (argc == 4) {
    FLAGS_batch_size = atoi(argv[3]);
  }
  xdims[0] = FLAGS_batch_size; // number of images
  rdims[0] = FLAGS_batch_size; // number of images

  // Load data into x and y
  float *x = allocate<float>(xdims);
  float *y = allocate<float>(rdims);
  loadData(x, y);

  // Load model
  float *conv1 = allocate<float>(conv1dims);
  float *conv2 = allocate<float>(conv2dims);
  float *fc1   = allocate<float>(fc1dims);
  float *fc2   = allocate<float>(fc2dims);
  loadModel(conv1, conv2, fc1, fc2);

  // Perform foward opertion
  int *out = zeros<int>(FLAGS_batch_size);

  // get start time
  const auto start = now();

  forward_operation(x, conv1, conv2, fc1, fc2, out);

  // get end time
  const auto end = now();

  // get elapsed time in milliseconds
  const auto elapsed =
      std::chrono::duration<double, std::milli>(end - start).count();

  // Get reference
  int *ref = zeros<int>(FLAGS_batch_size);
  argmax(y, rdims, ref);

  // Calculate correctness
  int num_correct = 0;
  for (const auto i : range(0, FLAGS_batch_size)) {
    if (out[i] == ref[i]) {
      num_correct++;
    }
  }
  std::cout << "Done with " << FLAGS_batch_size << " queries in "
            << "elapsed = " << elapsed << " milliseconds. Correctness: "
            << static_cast<float>(num_correct) / FLAGS_batch_size << "\n";

  delete[] x;
  delete[] y;
  delete[] conv1;
  delete[] conv2;
  delete[] fc1;
  delete[] fc2;
  delete[] out;
  delete[] ref;

  return 0;
}
